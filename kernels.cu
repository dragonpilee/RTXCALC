
#include <hip/hip_runtime.h>
extern "C" {

__global__ void add(float *a, float *b, float *c) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx < 1) c[idx] = a[idx] + b[idx];
}

__global__ void sub(float *a, float *b, float *c) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx < 1) c[idx] = a[idx] - b[idx];
}

__global__ void mul(float *a, float *b, float *c) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx < 1) c[idx] = a[idx] * b[idx];
}

__global__ void divide(float *a, float *b, float *c) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx < 1 && b[idx] != 0) c[idx] = a[idx] / b[idx];
}

}
